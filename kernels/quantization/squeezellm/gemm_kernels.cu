#include "hip/hip_runtime.h"
#include <torch/all.h>
#include <torch/python.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

// half tensor
#include <c10/cuda/CUDAStream.h>
#include <ATen/cuda/CUDATensorMethods.cuh>

// atomicAdd for double precision floating-point numbers on
// hardware with compute capability < 6.0
// https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#atomic-functions
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 600
__device__ doule atomicAdd(
    double* address,
    double val
) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(
            address_as_ull,
            assumed,
            __double_as_longlong(val + __longlong_as_double(assumed))
        );
    
    // NOTE: uses integer comparison to avoid hanging in case of NaN (NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif

__device__ inline unsigned int as_unsigned(int i) {
    return *reinterpret_cast<unsigned int*>(&i);
}

const int BLOCKWIDTH = 128;
const int BLOCKHEIGHT3 = 12;
const int BLOCKHEIGHT4 = 16;

__global__ void NUQ4MatMulKernel(
    const half2* __restrict__ vec,
    const int* __restrict__ mat,
    half2* __restrict__ mul,
    const __half* __restrict__ lookup_table,
    int height,
    int width,
    int batch,
    int vec_height
);

// 4bit matvec kernel (LUT-based)
void squeezellm_gemm(
    torch::Tensor vec,
    torch::Tensor mat,
    torch::Tensor mul,
    torch::Tensor lookup_table
) {
    int height = mat.size(0);
    int width = mat.size(1);

    int batch = vec.size(0);
    int vec_height = vec.size(1);

    dim3 blocks(
        (height + BLOCKHEIGHT4 - 1) / BLOCKHEIGHT4,
        (width + BLOCKWIDTH - 1) / BLOCKWIDTH
    );
    dim3 threads(BLOCKWIDTH);

    NUQ4MatMulKernel<<<blocks, threads>>>(
        (half2*) vec.data<at::Half>(),
        mat.data_ptr<int>(),
        (half2*) mul.data<at::Half>(),
        (__half*) lookup_table.data<at::Half>(),
        height, width, batch, vec_height
    );
}

// 4bit matvec kernel (LUT-based)
__global__ void NUQ4MatMulKernel(
    const  half2* __restrict__ vec,
    const    int* __restrict__ mat,
           half2* __restrict__ mul,
    const  __half* __restrict__ lookup_table,
    int height,
    int width,
    int batch,
    int vec_height
) {

  const int blockwidth2 = BLOCKWIDTH / 2;

  int row = BLOCKHEIGHT4 * blockIdx.x;
  int col =  BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ half2 blockvec[blockwidth2];

  __shared__ __half deq2[16][BLOCKWIDTH];
  int off = threadIdx.x;
  int column_offset = col * 16;
  for (int val = 0; val < 16; val += 1) {
    int lut_index = column_offset + val;
    deq2[val][off] = lookup_table[lut_index];
  }

  __half res;
  half2 res2;
  half2 tmp2;

  int i;
  int k;

  unsigned int tmp1;
  unsigned int lut_index1, lut_index2;

  for (int b = 0; b < batch; ++b){
    i = width * row + col;
    res = __int2half_rd(0);
    k = 0;

    __syncthreads();
    if (threadIdx.x < blockwidth2)
      blockvec[threadIdx.x] = vec[b * vec_height / 2 + (row / BLOCKHEIGHT4) * blockwidth2 + threadIdx.x];
    __syncthreads();

    while (k < blockwidth2) {
      tmp1 = as_unsigned(mat[i]);

      res2 = {};
      tmp2 = {};

      lut_index1 = tmp1 & 0xF;
      lut_index2 = (tmp1 >> 4) & 0xF;
      tmp2.x = deq2[lut_index1][off];
      tmp2.y = deq2[lut_index2][off];
      res2 = __hfma2(tmp2, blockvec[k + 0], res2);

      lut_index1 = (tmp1 >> 8) & 0xF;
      lut_index2 = (tmp1 >> 12) & 0xF;
      tmp2.x = deq2[lut_index1][off];
      tmp2.y = deq2[lut_index2][off];
      res2 = __hfma2(tmp2, blockvec[k + 1], res2);

      lut_index1 = (tmp1 >> 16) & 0xF;
      lut_index2 = (tmp1 >> 20) & 0xF;
      tmp2.x = deq2[lut_index1][off];
      tmp2.y = deq2[lut_index2][off];
      res2 = __hfma2(tmp2, blockvec[k + 2], res2);

      lut_index1 = (tmp1 >> 24) & 0xF;
      lut_index2 = (tmp1 >> 28) & 0xF;
      tmp2.x = deq2[lut_index1][off];
      tmp2.y = deq2[lut_index2][off];
      res2 = __hfma2(tmp2, blockvec[k + 3], res2);

      res = __hadd(__hadd(res2.x, res2.y), res);

      i += width;
      k += 4;
    }

    // col%2 -> only set one of the two values
    half2 res3 = {};
    if (col % 2 == 0) {
      res3.x = res;
    } else {
      res3.y = res;
    }

    atomicAdd(&mul[b * width / 2 + col / 2], res3);
  }
}